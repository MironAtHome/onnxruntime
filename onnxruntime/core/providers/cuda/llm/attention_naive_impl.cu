/*
Copyright (c) Microsoft Corporation.
Licensed under the MIT License.
*/

#include "core/providers/cuda/llm/attention_naive_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include <hip/hip_fp16.h>

using namespace onnxruntime::cuda;
using onnxruntime::concurrency::ThreadPool;

namespace onnxruntime {
namespace cuda {

Status GemmMatMul(
    hipStream_t stream, bool has_bias, bool has_scales,
    int32_t dtype_A, int32_t dtype_B,
    int32_t dtype_C, int32_t dtype_Y,
    bool trans_A, bool trans_B, const void* p_input_a, const void* p_input_b,
    const void* p_input_c, const void* p_scale_a, const void* p_scale_b,
    const void* p_scale_y, void* p_output_y, int M, int N, int K, int lda,
    int ldb, int ldd, bool row_major_compute, int64_t sm_count, int iepilogue,
    float alpha, float beta) {
  // TODO: Synchronization should be moved outside of this function.
  // TODO: The function should be split in two parts: create descriptors and run hipblasLtMatmul.
  hipblasLtEpilogue_t epilogue = static_cast<hipblasLtEpilogue_t>(iepilogue);
  CUDA_RETURN_IF_ERROR(hipStreamSynchronize(stream));

  hipblasLtHandle_t cublasLt;
  CUBLAS_RETURN_IF_ERROR(hipblasLtCreate(&cublasLt));

  hipblasLtMatmulDesc_t operationDesc = nullptr;
  hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr,
                         Ddesc = nullptr;

  // Create matrix descriptors. Not setting any extra attributes.
  hipDataType a_cuda_type = onnxruntime::cuda::ToCudaDataType(dtype_A);
  hipDataType b_cuda_type = onnxruntime::cuda::ToCudaDataType(dtype_B);
  hipDataType d_cuda_type = onnxruntime::cuda::ToCudaDataType(dtype_Y);
  hipDataType scale_cuda_type =
      onnxruntime::cuda::ToCudaDataType(ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT);
  hipDataType bias_cuda_type = onnxruntime::cuda::ToCudaDataType(dtype_C);

  hipblasComputeType_t compute_type;
  switch (d_cuda_type) {
    case HIP_R_16F:
      switch (a_cuda_type) {
#if !defined(DISABLE_FLOAT8_TYPES)
#if CUDA_VERSION < 11080
#error HIP_R_8F_E4M3 (float 8 types) is defined with CUDA>=11.8. Set flag DISABLE_FLOAT8_TYPES.
#endif
        case HIP_R_8F_E4M3:
        case HIP_R_8F_E5M2:
          compute_type = HIPBLAS_COMPUTE_32F;
          break;
#endif
        default:
          compute_type = HIPBLAS_COMPUTE_32F_FAST_16F;
          break;
      }
      break;
    case HIP_R_16BF:
      compute_type = HIPBLAS_COMPUTE_32F_FAST_16BF;
      break;
    case HIP_R_32F:
      compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;
      break;
    default:
      ORT_THROW("Unable to determine computeType in operator GemmFloat8.");
  }

  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(
      &Adesc, a_cuda_type, trans_A ? K : M, trans_A ? M : K, lda));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(
      &Bdesc, b_cuda_type, trans_B ? N : K, trans_B ? K : N, ldb));
  CUBLAS_RETURN_IF_ERROR(
      hipblasLtMatrixLayoutCreate(&Ddesc, d_cuda_type, M, N, ldd));

  if (row_major_compute) {
    hipblasLtOrder_t matrixOrder = HIPBLASLT_ORDER_ROW;
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
  }

  CUBLAS_RETURN_IF_ERROR(
      hipblasLtMatmulDescCreate(&operationDesc, compute_type, scale_cuda_type));
  hipblasOperation_t ctransa = trans_A ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t ctransb = trans_B ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &ctransa, sizeof(ctransa)));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &ctransb, sizeof(ctransb)));

#if CUDA_VERSION >= 11060
  // CUBLASLT_MATMUL_DESC_SM_COUNT_TARGET exists from https://docs.nvidia.com/cuda/archive/11.6.0/pdf/CUBLAS_Library.pdf
  if (sm_count != 0) {
    int math_sm_count = static_cast<int>(sm_count);
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, CUBLASLT_MATMUL_DESC_SM_COUNT_TARGET, &math_sm_count,
        sizeof(math_sm_count)));
  }
#endif

  if (has_scales) {
    // gemm float 8
#if CUDA_VERSION >= 11080
    // CUBLASLT_MATMUL_DESC_FAST_ACCUM, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER,
    // HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER exist from https://docs.nvidia.com/cuda/archive/11.8.0/pdf/CUBLAS_Library.pdf
    const int8_t ifast_accumulation_mode = 1;
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc,
        hipblasLtMatmulDescAttributes_t::CUBLASLT_MATMUL_DESC_FAST_ACCUM,
        &ifast_accumulation_mode, sizeof(ifast_accumulation_mode)));
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &p_scale_a,
        sizeof(p_scale_a)));
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &p_scale_b,
        sizeof(p_scale_b)));
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &p_scale_y,
        sizeof(p_scale_b)));
#endif

    // float 8
#if !defined(DISABLE_FLOAT8_TYPES)
    if (dtype_Y == ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT8E4M3FN ||
        dtype_Y == ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT8E5M2) {
      // For FP8 output, cuBLAS requires C_type to be same as bias_type
      CUBLAS_RETURN_IF_ERROR(
          hipblasLtMatrixLayoutCreate(&Cdesc, bias_cuda_type, M, N, ldd));
      CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
          operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_DATA_TYPE, &bias_cuda_type,
          sizeof(bias_cuda_type)));
    } else {
      CUBLAS_RETURN_IF_ERROR(
          hipblasLtMatrixLayoutCreate(&Cdesc, d_cuda_type, M, N, ldd));
    }
#else
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutCreate(&Cdesc, d_cuda_type, M, N, ldd));
#endif
  } else {
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutCreate(&Cdesc, d_cuda_type, M, N, ldd));
  }

  if (row_major_compute) {
    hipblasLtOrder_t matrixOrder = HIPBLASLT_ORDER_ROW;
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Ddesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
  }

  hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE,
                                 &epilogue, sizeof(epilogue));

  // See
  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulPreferenceAttributes_t#cublasltmatmulpreferenceattributes-t
  // The workspace should be allocated once from OpKernelContext assuming
  // only one cuda function is running at a time (which is not necessarily true
  // with H100).
  size_t workspaceSize = static_cast<size_t>(1 << 25);  // suggested fixed value 32Mb
  hipblasLtMatmulPreference_t preference = nullptr;
  hipblasLtMatmulPreferenceCreate(&preference);
  hipblasLtMatmulPreferenceSetAttribute(preference,
                                       HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                       &workspaceSize, sizeof(workspaceSize));

  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulAlgoGetHeuristic#cublasltmatmulalgogetheuristic
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  int returnedResults = 0;
  hipblasStatus_t cuda_status = hipblasLtMatmulAlgoGetHeuristic(
      cublasLt, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1,
      &heuristicResult, &returnedResults);
  ORT_ENFORCE(
      returnedResults > 0 && cuda_status == HIPBLAS_STATUS_SUCCESS,
      " Unable to find any suitable algorithm due to ",
      onnxruntime::cuda::cublasGetErrorEnum(cuda_status),
      ", returnedResults=", returnedResults,
      ", alpha=", alpha, ", beta=", beta,
      ", A_type=", onnxruntime::cuda::CudaDataTypeToString(a_cuda_type),
      ", B_type=", onnxruntime::cuda::CudaDataTypeToString(b_cuda_type),
      ", C_type=", onnxruntime::cuda::CudaDataTypeToString(bias_cuda_type),
      ", result_type=", onnxruntime::cuda::CudaDataTypeToString(d_cuda_type),
      ", bias_type=", onnxruntime::cuda::CudaDataTypeToString(bias_cuda_type),
      ", scale_type=", onnxruntime::cuda::CudaDataTypeToString(scale_cuda_type),
      ", computeType=", onnxruntime::cuda::CublasComputeTypeToString(compute_type),
      ", epilogue=", epilogue, ", smCount=", sm_count, ", transA=", trans_A,
      ", transB=", trans_B,
      ", fastAccumulationMode=", 1,
      ", M=", M, ", N=", N, ", K=", K,
      ", lda=", lda, ", ldb=", ldb, ", ldd=", ldd,
      ", workspaceSize=", workspaceSize, ", rowMajorCompute=", (row_major_compute ? 1 : 0),
      ". Check NVIDIA documentation to see what combination is valid: ",
      "https://docs.nvidia.com/cuda/cublas/"
      "index.html?highlight=hipblasLtMatmulAlgoGetHeuristic#"
      "cublasltmatmulalgogetheuristic. CUDA>=11.8 is required to use float 8 types.");

  void* workspace = nullptr;
  if (workspaceSize > 0) {
    CUDA_RETURN_IF_ERROR(hipMalloc(reinterpret_cast<void**>(&workspace), workspaceSize));
  }
  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmul#cublasltmatmul
  const void* bias = has_bias ? p_input_c : p_output_y;
  cuda_status = hipblasLtMatmul(
      cublasLt, operationDesc, static_cast<const void*>(&alpha), /* alpha */
      p_input_a,                                                 /* A */
      Adesc, p_input_b,                                          /* B */
      Bdesc, static_cast<const void*>(&beta),                    /* beta */
      bias,                                                      /* C */
      Cdesc, p_output_y,                                         /* Y */
      Ddesc, &heuristicResult.algo,                              /* algo */
      workspace,                                                 /* workspace */
      workspaceSize, stream);                                    /* stream */
  ORT_ENFORCE(
      cuda_status == HIPBLAS_STATUS_SUCCESS,
      " Unable to run hipblasLtMatmul due to ",
      onnxruntime::cuda::cublasGetErrorEnum(cuda_status),
      ", returnedResults=", returnedResults, ", alpha=", alpha,
      ", A_type=", onnxruntime::cuda::CudaDataTypeToString(a_cuda_type),
      ", B_type=", onnxruntime::cuda::CudaDataTypeToString(b_cuda_type),
      ", result_type=", onnxruntime::cuda::CudaDataTypeToString(d_cuda_type),
      ", bias_type=", onnxruntime::cuda::CudaDataTypeToString(bias_cuda_type),
      ", scale_type=", onnxruntime::cuda::CudaDataTypeToString(scale_cuda_type),
      ", computeType=", onnxruntime::cuda::CublasComputeTypeToString(compute_type),
      ", epilogue=", epilogue, ", smCount=", sm_count, ", transA=", trans_A,
      ", transB=", trans_B,
      ", fastAccumulationMode=", 1,
      " M=", M, " N=", N, ", K=", K, ", lda=", lda, ", ldb=",
      ldb, ", ldd=", ldd, ", workspaceSize=", workspaceSize,
      ", rowMajorCompute=", (row_major_compute ? 1 : 0),
      ". CUDA>=11.8 is required to use float 8 types.");

  if (workspaceSize > 0) {
    CUDA_RETURN_IF_ERROR(hipFree(workspace));
  }

  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceDestroy(preference));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Ddesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Cdesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Bdesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Adesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescDestroy(operationDesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtDestroy(cublasLt));
  return Status::OK();
}

}  // namespace cuda
}  // namespace onnxruntime
